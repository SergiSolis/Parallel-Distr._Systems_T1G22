
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>

#define N 1024
#define ROWSIZE 9


__global__ void cuspmv(int offset, int nsize, double* dvals, int *dcols, double* dx, double *dy)
{


}


void spmv_cpu(int offset, int nsize, double* vals, int* cols, double* x, double* y)
{
	int end_offset = offset + nsize;
		for(int i = offset; i < end_offset; i++){
			for(int j = 0; j < ROWSIZE; j++){
				y[i] += vals[(ROWSIZE*i) + j] * x[cols[(ROWSIZE*i) + j]];
			}
		}
}


void fill_matrix(double* vals, int* cols)
{

    int indx[ROWSIZE];
    int row_count=0;
    for(int j=0; j< N ; j++){
        for(int i=0; i<N; i++){

            indx[0] = i     + (j - 2)*N;
            indx[1] = i     + (j - 1)*N;
            indx[2] = i - 2 +       j*N;
            indx[3] = i - 1 +       j*N;
            indx[4] = i + j*N;
            indx[5] = i + 1 +       j*N;
            indx[6] = i + 2 +       j*N;
            indx[7] = i    +  (j + 1)*N;
            indx[8] = i    +  (j + 2)*N;


            for(int row=0; row < ROWSIZE; row++)
            {
                if(indx[row] < 0 || indx[row] >= N*N)
                {
                    cols[row + row_count*ROWSIZE] = i + j*N;
                    vals[row + row_count*ROWSIZE] = 0.0;
                }
                else
                {
                    if(row == 4) {
                        cols[row + row_count*ROWSIZE] = indx[row];
                        vals[row + row_count*ROWSIZE] = 0.95;
                    }
                    else
                    {
                        cols[row + row_count*ROWSIZE] = indx[row];
                        vals[row + row_count*ROWSIZE] = -0.95/(ROWSIZE - 1);
                    }
                }
            }
            row_count++;
        }
    }

    vals[4 + (N*N/2 + N/2)*ROWSIZE] =  1.001*vals[4 + (N*N/2 + N/2)*ROWSIZE];
}

int main()
{


    double* x;
    double* y_cpu;
    double* y_gpu;

    double* Avals;
    int*    Acols;

    int     vec_size = N*N;
    int     offset;

    double  norm2=0.0;


    x = (double*) malloc (vec_size*sizeof(double));
    y_cpu = (double*) malloc (vec_size*sizeof(double));
    y_gpu = (double*) malloc (vec_size*sizeof(double));
    Avals = (double*) malloc (ROWSIZE*vec_size*sizeof(double));
    Acols = (int*) malloc (ROWSIZE*vec_size*sizeof(int));



    //Filling the vector with a sinusoidal for validating the subroutine
    for(int i = 0; i < vec_size; i++)
    {
        x[i] = sin(i*0.01);
        y_cpu[i] = 0.0;
        y_gpu[i] = 0.0;
    }


    fill_matrix(Avals, Acols);

    offset=0;
    spmv_cpu(offset, vec_size, Avals, Acols, x, y_cpu);


    // Here you must start your CUDA code
    double* dx;
    double* dy_gpu;

    double* dAvals;
    int*    dAcols;


    // Allocate arrays in GPU
	
	hipMalloc(&dx,vec_size*sizeof(double));
	hipMalloc(&dy_gpu,vec_size*sizeof(double));
	hipMalloc(&dAvals,vec_size*ROWSIZE*sizeof(double));
	hipMalloc(&dAcols,vec_size*ROWSIZE*sizeof(int));

    // Transfer data to GPU
	
	hipMemcpy(dx,x,vec_size*sizeof(double),hipMemcpyHostToDevice); //segundo argumento creo que esta mal de todos
    hipMemcpy(dy_gpu,y_gpu,vec_size*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(dAvals,Avals,vec_size*ROWSIZE*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dAcols,Acols,vec_size*ROWSIZE*sizeof(int),hipMemcpyHostToDevice);

    //Calculate threads and blocks
	int threads;
	int blocks;

	threads = 512;
	blocks = N;

    //Create the gridBlock
	dim3 gridBlocks(blocks,1,1);
	dim3 gridThreads(threads,1,1);
	
    for( int i=0; i<100; i++){
        //call your GPU kernel here
        

    }

    // Transfer your result back
	
	hipMemcpy(y_gpu,dy_gpu,vec_size*sizeof(double),hipMemcpyDeviceToHost);
	
    // Free arrays in GPU
	
	hipFree(dx);
	hipFree(dy_gpu);
	hipFree(dAvals);
	hipFree(dAcols);
	


    //Comparison between gpu and cpu results
    norm2=0.0;
    for(int i=0; i<vec_size; i++)
        norm2 += (y_cpu[i]-y_gpu[i])*(y_cpu[i]-y_gpu[i]);

    norm2= sqrt(norm2);

    printf("spmv comparison cpu vs gpu error: %e, offset %d, size %d\n", norm2, offset, vec_size);


}
