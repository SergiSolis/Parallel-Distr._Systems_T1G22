
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>

#define N 1024
#define ROWSIZE 9


__global__ void cuspmv(int offset, int nsize, double* dvals, int *dcols, double* dx, double *dy)
{


}


void spmv_cpu(int offset, int nsize, double* vals, int* cols, double* x, double* y)
{

}


void fill_matrix(double* vals, int* cols)
{

    int indx[ROWSIZE];
    int row_count=0;
    for(int j=0; j< N ; j++){
        for(int i=0; i<N; i++){

            indx[0] = i     + (j - 2)*N;
            indx[1] = i     + (j - 1)*N;
            indx[2] = i - 2 +       j*N;
            indx[3] = i - 1 +       j*N;
            indx[4] = i + j*N;
            indx[5] = i + 1 +       j*N;
            indx[6] = i + 2 +       j*N;
            indx[7] = i    +  (j + 1)*N;
            indx[8] = i    +  (j + 2)*N;


            for(int row=0; row < ROWSIZE; row++)
            {
                if(indx[row] < 0 || indx[row] >= N*N)
                {
                    cols[row + row_count*ROWSIZE] = i + j*N;
                    vals[row + row_count*ROWSIZE] = 0.0;
                }
                else
                {
                    if(row == 4) {
                        cols[row + row_count*ROWSIZE] = indx[row];
                        vals[row + row_count*ROWSIZE] = 0.95;
                    }
                    else
                    {
                        cols[row + row_count*ROWSIZE] = indx[row];
                        vals[row + row_count*ROWSIZE] = -0.95/(ROWSIZE - 1);
                    }
                }
            }
            row_count++;
        }
    }

    vals[4 + (N*N/2 + N/2)*ROWSIZE] =  1.001*vals[4 + (N*N/2 + N/2)*ROWSIZE];
}

int main()
{


    double* x;
    double* y_cpu;
    double* y_gpu;

    double* Avals;
    int*    Acols;

    int     vec_size = N*N;
    int     offset;

    double  norm2=0.0;


    x = (double*) malloc (vec_size*sizeof(double));
    y_cpu = (double*) malloc (vec_size*sizeof(double));
    y_gpu = (double*) malloc (vec_size*sizeof(double));
    Avals = (double*) malloc (ROWSIZE*vec_size*sizeof(double));
    Acols = (int*) malloc (ROWSIZE*vec_size*sizeof(int));



    //Filling the vector with a sinusoidal for validating the subroutine
    for(int i = 0; i < vec_size; i++)
    {
        x[i] = sin(i*0.01);
        y_cpu[i] = 0.0;
        y_gpu[i] = 0.0;
    }


    fill_matrix(Avals, Acols);

    offset=0;
    spmv_cpu(offset, vec_size, Avals, Acols, x, y_cpu);


    // Here you must start your CUDA code
    double* dx;
    double* dy_gpu;

    double* dAvals;
    int*    dAcols;


    // Allocate arrays in GPU


    // Transfer data to GPU


    //Calculate threads and blocks


    //Create the gridBlock

    for( int i=0; i<100; i++){
        //call your GPU kernel here
        

    }

    // Transfer your result back

    // Free arrays in GPU



    //Comparison between gpu and cpu results
    norm2=0.0;
    for(int i=0; i<vec_size; i++)
        norm2 += (y_cpu[i]-y_gpu[i])*(y_cpu[i]-y_gpu[i]);

    norm2= sqrt(norm2);

    printf("spmv comparison cpu vs gpu error: %e, offset %d, size %d\n", norm2, offset, vec_size);


}